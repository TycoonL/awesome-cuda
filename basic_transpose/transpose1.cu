#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void transpose(float* in, float* out, int width, int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		int index_in = y * width + x;
		int index_out = x * height + y;
		//out[index_in] = __ldg(&in[index_out]);//Execution time: 0.019456 ms
		//out[index_out] = __ldg(&in[index_in]);//Execution time: 0.021504 ms
		//out[index_in] = in[index_out];//out�ϲ�д�룬in�Ǻϲ�����  Execution time: 0.019456 ms
		out[index_out] = in[index_in];//in�ϲ����ʣ�out�Ǻϲ�д��  Execution time: 0.021504 ms
	}
}

int main() {
	int width =64;
	int height = 64;

	int size = width * height * sizeof(float);
	float* in = (float*)malloc(size);
	float* out = (float*)malloc(size);

	for (int i = 0; i < width * height; i++) {
		in[i] = (float)i;
	}

	float *d_in, *d_out;
	hipMalloc(&d_in, size);
	hipMalloc(&d_out, size);

	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 block_size(32, 32);
	dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);
	hipEventRecord(start);
	transpose << <grid_size, block_size >> > (d_in, d_out, width, height);
	hipEventRecord(stop);

	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Execution time: %f ms\n", milliseconds);

	hipFree(d_in);
	hipFree(d_out);
	free(in);
	free(out);

	return 0;
}
