#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"   // ���� CUDA ����ʱ���ͷ�ļ�
#include ""   // ���� CUDA �豸������ִ�в�����ͷ�ļ�
#include "iostream"

//�����豸��̬ȫ�ֱ���
__device__ int cuda_x = 1;
__device__ int cuda_y[2];

__global__
void kernel()
{
	cuda_y[0] += cuda_x;
	cuda_y[1] += cuda_x;
	printf("cuda_y1=%d,cuda_y2=%d\n", cuda_y[0], cuda_y[1]);
}

int main()
{

	int host[2] = {10, 20};
	//hipMemcpy(cuda_y,host, sizeof(int) * 2,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_y), host, sizeof(int) * 2);//
	kernel << <1, 1 >> > ();
	hipDeviceSynchronize();
	hipMemcpyFromSymbol(host, HIP_SYMBOL(cuda_y), sizeof(int) * 2);
	printf("host1=%d,host2=%d\n", host[0], host[1]);

	while (1);   // �������ǰ�������У��Ա�鿴��������
	return 0;
}