#include "hip/hip_runtime.h"

#include "iostream"


//核函数
__global__ void hello_from_device()
{
	int bid_x = blockIdx.x;
	int bid_y = blockIdx.y;
	int tid = threadIdx.x;
	printf("hello world from block (%d,%d) and thread %d\n", bid_x,bid_y, tid);
}


int main(void)
{
	dim3 gridsize(2, 2);
	hello_from_device<<<gridsize,3 >>>();
	hipDeviceSynchronize();//同步主机与设备
	return 0;
}
