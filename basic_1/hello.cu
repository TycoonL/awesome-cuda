#include "hip/hip_runtime.h"

#include "iostream"

//打印GPU设备信息
void print_GPU_device_info()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; i++)
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
		std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
		std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
		std::cout << "每个SM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
		std::cout << "每个SM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
		std::cout << "每个Block的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
		std::cout << "每个Block的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
		std::cout << "每个Block中可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
		std::cout << "======================================================" << std::endl;

	}
}


int main()
{
	print_GPU_device_info();

	while (1);
	return 0;
}
